#include <stdio.h>
#include <hip/hip_runtime.h>

// kernel
__global__ void square(float* d_out, float* d_in){
  int idx = threadIdx.x;
  float f = d_in[idx];
  d_out[idx] =  f * f;
}

int main(int argc, char** argv){
  const int ARRAY_SIZE = 64;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
  
  // generate the input array on the host
  float h_in[ARRAY_SIZE];
  for(int i = 0; i < ARRAY_SIZE; i++){
    h_in[i] = float(i);
  }
  
  float h_out[ARRAY_SIZE];

  // declare GPU mem pointers
  float* d_in;
  float* d_out;

  // allocate GPU mem
  hipMalloc((void **) &d_in, ARRAY_BYTES); 
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // transer the array to the gpu
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // launch the kernel 
  square<<<1, ARRAY_SIZE>>>(d_out, d_in);

  // copy back the result array to the gpu
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  for(int i = 0; i < ARRAY_SIZE; i++){
    printf("%f", h_out[i]);
    printf(((i%4) != 3) ? "\t" : "\n");
  }

  // free gpue memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;

}
